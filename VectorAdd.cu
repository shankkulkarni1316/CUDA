// sorting
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrixmult (int *gpu_a, int *gpu_c, int N) {

	int j, x;
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N) {
	x = 0;
	for (j = 0; j < N; j++) {  /* count number less than it */
		if (gpu_a[tid] > gpu_a[j]) x++;
				
	}
	gpu_c[x] = gpu_a[tid];
	}

}


void cpu_matrixmult(int *cpu_a, int *cpu_d, int N) {
	int i, j, k;
	int x;
	

	for (k = 0; k < N; k++) {      /* for each number */
		x = 0;
		for (j = 0; j < N; j++) {     /* count number less than it */
			if (cpu_a[k] > cpu_a[j] ) x++;
			
		}
		cpu_d[x] = cpu_a[k];	 /* copy number into correct place */
		
	}
	


}



int main(int argc, char *argv[]) {
	int i, j; 							// loop counters

	
	int N, B, T;  						// size of array in each dimension
	int *a,*c,*d;
	int *dev_a, *dev_c;
	
	int size;							// number of bytes in arrays
	hipEvent_t start, stop;     				// using cuda events to measure time
	float elapsed_time_ms1, elapsed_time_ms2;       	// which is applicable for asynchronous code also
	float speedup;
	hipEventCreate(&start);		
	hipEventCreate(&stop);
	
	

	
	

	
	printf("Enter number of threads in a block:\n");
	scanf("%d", &T);
	if (T > 1024) {  // check for maximum value of T
		printf("Maximum number of threads per block can be 1024. Hence T will be set to its maximum that is: 1024\n");
		T = 1024;
	}
	
	printf("Enter number of blocks in a grid:\n");
	scanf("%d", &B);
	if (B > 65535) {  // check for maximum value of B
		printf("Maximum number of blocks in a grid can be 65535. Hence B will be set to its maximum that is: 65535\n");
		B = 65535;
	}	
	
	printf("Enter number of random numbers N:\n");
	scanf("%d", &N);	// keyboard input



	size = N * sizeof(int);				// number of bytes in total in arrays

	a = (int*) malloc(size);					//dynamically allocated memory for arrays on host
	
	c = (int*) malloc(size);					// results from GPU
	d = (int*) malloc(size);				// results from CPU

	hipMalloc((void**)&dev_a, size);			// allocate memory on device
	
	hipMalloc((void**)&dev_c, size);
		
		
	
	
	


		

	srand(3); //initialize random number generator
	for (i=0; i < N; i++) { //load array with numbers
		a[i] = (int)rand();
	}
		
	/*	
		printf("Vector A is:\n");
		for(i=0; i < N; i++) {
			printf("\t%d", a[i]);
		}
		printf("\n");
	*/
	
		
	/* ------------- COMPUTATION DONE ON GPU ----------------------------*/



		hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
		

		hipEventRecord(start, 0); 			// here start time, after memcpy

		gpu_matrixmult<<<1,T*B>>>(dev_a,dev_c,N);
		hipMemcpy(c, dev_c, size , hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);     			// measure end time
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time_ms1, start, stop );

		printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms1); 
	/* ------------- COMPUTATION DONE ON HOST CPU ----------------------------*/

		hipEventRecord(start, 0);			// use same timing*

		cpu_matrixmult(a,d,N);				// do calculation on host

		hipEventRecord(stop, 0);     		// measure end time
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time_ms2, start, stop );

		printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms2);  // 

		speedup = elapsed_time_ms2/elapsed_time_ms1;
		printf("speed-up is: %f\n", speedup);

		/*
		printf("Vector C is:\n");
		for(i=0; i < N; i++) {
			printf("\t%d", c[i]);
		}
		printf("\n");
		
		printf("Vector D is:\n");
		for(i=0; i < N; i++) {
			printf("\t%d", d[i]);
		}
		printf("\n");	

		*/
	
	
		// checking if both methods give same answer
		int error = 0;
		for (i=0; i < N; i++) {
			
			if (c[i] != d[i]) error = -1;
			
		}
		if (error == -1) printf("ERROR, sequential and parallel versions give different answers\n");
		else printf("Sequential and parallel versions give same answers\n");

		printf("values of T, B and N are: %d\t%d\t%d\n", T, B, N);
	
		
	
	/* --------------  clean up  ---------------------------------------*/
	free(a); free(c);
	hipFree(dev_a);
	
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	
	
	
	return 0;
}